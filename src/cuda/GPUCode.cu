#include "hip/hip_runtime.h"
#include "GPUCode.cuh"

#include <cpu/Vec3.h>
#include <cpu/Sphere.h>
#include <vector>

// TODO: Find out why it's required for cuda_gl_interop
#include <GL/glew.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

#include <cassert>

struct GPUContext
{
    bool init;
    hipGraphicsResource_t pixelBuffer;
};

//BGRA texture format
__global__ void RenderKernel(uchar4* renderTarget, int width) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y; 
    renderTarget[y * width + x] = make_uchar4(x, y, x + y, 255);
}


void CreateGPUContext(GPUContext** context) {
    *context = (GPUContext*) malloc(sizeof(GPUContext));
    memset(*context, 0, sizeof(GPUContext));
}

void FreeGPUContext(GPUContext* context) {
    free(context);
}

void InitGPURendering(GPUContext* context) {
    hipSetDevice(0);
    context->init = true;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
}

void CopyImageToGPU(GPUContext* context, uint8_t* pixels, int width, int height) {
    hipError_t error = hipGraphicsMapResources(1, &context->pixelBuffer, 0);
    assert(error == hipSuccess);

    // Map buffer object
    uchar4* renderTarget = 0;
    size_t num_bytes;
    error = hipGraphicsResourceGetMappedPointer((void**)&renderTarget, &num_bytes, context->pixelBuffer);
    assert(renderTarget);

    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks(width / threadsPerBlock.x, height / threadsPerBlock.y);
    RenderKernel<<<numBlocks, threadsPerBlock>>>(renderTarget, width);

    // Unmap buffer object
    error = hipGraphicsUnmapResources(1, &context->pixelBuffer, 0);
    assert(error == hipSuccess);
}

void RegisterPixelBuffer(GPUContext* context, GLuint buffer) {
    hipError_t error = hipGraphicsGLRegisterBuffer(&context->pixelBuffer, buffer,
        cudaGraphicsMapFlagsWriteDiscard);
    assert(error == hipSuccess);
}

void UnregisterPixelBuffer(GPUContext* context) {
    hipGraphicsUnregisterResource(context->pixelBuffer);
}